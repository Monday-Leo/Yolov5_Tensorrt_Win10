#include "hip/hip_runtime.h"
#include <assert.h>
#include <vector>
#include <iostream>
#include "yololayer.h"
#include "cuda_utils.h"

namespace Tn
{
    template<typename T> 
    void write(char*& buffer, const T& val)
    {
        *reinterpret_cast<T*>(buffer) = val;
        buffer += sizeof(T);
    }

    template<typename T> 
    void read(const char*& buffer, T& val)
    {
        val = *reinterpret_cast<const T*>(buffer);
        buffer += sizeof(T);
    }
}

using namespace Yolo;

namespace nvinfer1
{
    YoloLayerPlugin::YoloLayerPlugin(int classCount, int netWidth, int netHeight, int maxOut, const std::vector<Yolo::YoloKernel>& vYoloKernel)
    {
        mClassCount = classCount;
        mYoloV5NetWidth = netWidth;
        mYoloV5NetHeight = netHeight;
        mMaxOutObject = maxOut;
        mYoloKernel = vYoloKernel;
        mKernelCount = vYoloKernel.size();

        CUDA_CHECK(hipHostMalloc(&mAnchor, mKernelCount * sizeof(void*)));
        size_t AnchorLen = sizeof(float)* CHECK_COUNT * 2;
        for (int ii = 0; ii < mKernelCount; ii++)
        {
            CUDA_CHECK(hipMalloc(&mAnchor[ii], AnchorLen));
            const auto& yolo = mYoloKernel[ii];
            CUDA_CHECK(hipMemcpy(mAnchor[ii], yolo.anchors, AnchorLen, hipMemcpyHostToDevice));
        }
    }
    YoloLayerPlugin::~YoloLayerPlugin()
    {
        for (int ii = 0; ii < mKernelCount; ii++)
        {
            CUDA_CHECK(hipFree(mAnchor[ii]));
        }
        CUDA_CHECK(hipHostFree(mAnchor));
    }

    // create the plugin at runtime from a byte stream
    YoloLayerPlugin::YoloLayerPlugin(const void* data, size_t length)
    {
        using namespace Tn;
        const char *d = reinterpret_cast<const char *>(data), *a = d;
        read(d, mClassCount);
        read(d, mThreadCount);
        read(d, mKernelCount);
        read(d, mYoloV5NetWidth);
        read(d, mYoloV5NetHeight);
        read(d, mMaxOutObject);
        mYoloKernel.resize(mKernelCount);
        auto kernelSize = mKernelCount * sizeof(YoloKernel);
        memcpy(mYoloKernel.data(), d, kernelSize);
        d += kernelSize;
        CUDA_CHECK(hipHostMalloc(&mAnchor, mKernelCount * sizeof(void*)));
        size_t AnchorLen = sizeof(float)* CHECK_COUNT * 2;
        for (int ii = 0; ii < mKernelCount; ii++)
        {
            CUDA_CHECK(hipMalloc(&mAnchor[ii], AnchorLen));
            const auto& yolo = mYoloKernel[ii];
            CUDA_CHECK(hipMemcpy(mAnchor[ii], yolo.anchors, AnchorLen, hipMemcpyHostToDevice));
        }
        assert(d == a + length);
    }

    void YoloLayerPlugin::serialize(void* buffer) const TRT_NOEXCEPT
    {
        using namespace Tn;
        char* d = static_cast<char*>(buffer), *a = d;
        write(d, mClassCount);
        write(d, mThreadCount);
        write(d, mKernelCount);
        write(d, mYoloV5NetWidth);
        write(d, mYoloV5NetHeight);
        write(d, mMaxOutObject);
        auto kernelSize = mKernelCount * sizeof(YoloKernel);
        memcpy(d, mYoloKernel.data(), kernelSize);
        d += kernelSize;

        assert(d == a + getSerializationSize());
    }

    size_t YoloLayerPlugin::getSerializationSize() const TRT_NOEXCEPT
    {
        return sizeof(mClassCount) + sizeof(mThreadCount) + sizeof(mKernelCount) + sizeof(Yolo::YoloKernel) * mYoloKernel.size() + sizeof(mYoloV5NetWidth) + sizeof(mYoloV5NetHeight) + sizeof(mMaxOutObject);
    }

    int YoloLayerPlugin::initialize() TRT_NOEXCEPT
    {
        return 0;
    }

    Dims YoloLayerPlugin::getOutputDimensions(int index, const Dims* inputs, int nbInputDims) TRT_NOEXCEPT
    {
        //output the result to channel
        int totalsize = mMaxOutObject * sizeof(Detection) / sizeof(float);

        return Dims3(totalsize + 1, 1, 1);
    }

    // Set plugin namespace
    void YoloLayerPlugin::setPluginNamespace(const char* pluginNamespace) TRT_NOEXCEPT
    {
        mPluginNamespace = pluginNamespace;
    }

    const char* YoloLayerPlugin::getPluginNamespace() const TRT_NOEXCEPT
    {
        return mPluginNamespace;
    }

    // Return the DataType of the plugin output at the requested index
    DataType YoloLayerPlugin::getOutputDataType(int index, const nvinfer1::DataType* inputTypes, int nbInputs) const TRT_NOEXCEPT
    {
        return DataType::kFLOAT;
    }

    // Return true if output tensor is broadcast across a batch.
    bool YoloLayerPlugin::isOutputBroadcastAcrossBatch(int outputIndex, const bool* inputIsBroadcasted, int nbInputs) const TRT_NOEXCEPT
    {
        return false;
    }

    // Return true if plugin can use input that is broadcast across batch without replication.
    bool YoloLayerPlugin::canBroadcastInputAcrossBatch(int inputIndex) const TRT_NOEXCEPT
    {
        return false;
    }

    void YoloLayerPlugin::configurePlugin(const PluginTensorDesc* in, int nbInput, const PluginTensorDesc* out, int nbOutput) TRT_NOEXCEPT
    {
    }

    // Attach the plugin object to an execution context and grant the plugin the access to some context resource.
    void YoloLayerPlugin::attachToContext(cudnnContext* cudnnContext, cublasContext* cublasContext, IGpuAllocator* gpuAllocator) TRT_NOEXCEPT
    {
    }

    // Detach the plugin object from its execution context.
    void YoloLayerPlugin::detachFromContext() TRT_NOEXCEPT {}

    const char* YoloLayerPlugin::getPluginType() const TRT_NOEXCEPT
    {
        return "YoloLayer_TRT";
    }

    const char* YoloLayerPlugin::getPluginVersion() const TRT_NOEXCEPT
    {
        return "1";
    }

    void YoloLayerPlugin::destroy() TRT_NOEXCEPT
    {
        delete this;
    }

    // Clone the plugin
    IPluginV2IOExt* YoloLayerPlugin::clone() const TRT_NOEXCEPT
    {
        YoloLayerPlugin* p = new YoloLayerPlugin(mClassCount, mYoloV5NetWidth, mYoloV5NetHeight, mMaxOutObject, mYoloKernel);
        p->setPluginNamespace(mPluginNamespace);
        return p;
    }

    __device__ float Logist(float data) { return 1.0f / (1.0f + expf(-data)); };

    __global__ void CalDetection(const float *input, float *output, int noElements,
        const int netwidth, const int netheight, int maxoutobject, int yoloWidth, int yoloHeight, const float anchors[CHECK_COUNT * 2], int classes, int outputElem)
    {

        int idx = threadIdx.x + blockDim.x * blockIdx.x;
        if (idx >= noElements) return;

        int total_grid = yoloWidth * yoloHeight;
        int bnIdx = idx / total_grid;
        idx = idx - total_grid * bnIdx;
        int info_len_i = 5 + classes;
        const float* curInput = input + bnIdx * (info_len_i * total_grid * CHECK_COUNT);

        for (int k = 0; k < CHECK_COUNT; ++k) {
            float box_prob = Logist(curInput[idx + k * info_len_i * total_grid + 4 * total_grid]);
            if (box_prob < IGNORE_THRESH) continue;
            int class_id = 0;
            float max_cls_prob = 0.0;
            for (int i = 5; i < info_len_i; ++i) {
                float p = Logist(curInput[idx + k * info_len_i * total_grid + i * total_grid]);
                if (p > max_cls_prob) {
                    max_cls_prob = p;
                    class_id = i - 5;
                }
            }
            float *res_count = output + bnIdx * outputElem;
            int count = (int)atomicAdd(res_count, 1);
            if (count >= maxoutobject) return;
            char *data = (char*)res_count + sizeof(float) + count * sizeof(Detection);
            Detection *det = (Detection*)(data);

            int row = idx / yoloWidth;
            int col = idx % yoloWidth;

            //Location
            // pytorch:
            //  y = x[i].sigmoid()
            //  y[..., 0:2] = (y[..., 0:2] * 2. - 0.5 + self.grid[i].to(x[i].device)) * self.stride[i]  # xy
            //  y[..., 2:4] = (y[..., 2:4] * 2) ** 2 * self.anchor_grid[i]  # wh
            //  X: (sigmoid(tx) + cx)/FeaturemapW *  netwidth
            det->bbox[0] = (col - 0.5f + 2.0f * Logist(curInput[idx + k * info_len_i * total_grid + 0 * total_grid])) * netwidth / yoloWidth;
            det->bbox[1] = (row - 0.5f + 2.0f * Logist(curInput[idx + k * info_len_i * total_grid + 1 * total_grid])) * netheight / yoloHeight;

            // W: (Pw * e^tw) / FeaturemapW * netwidth
            // v5: https://github.com/ultralytics/yolov5/issues/471
            det->bbox[2] = 2.0f * Logist(curInput[idx + k * info_len_i * total_grid + 2 * total_grid]);
            det->bbox[2] = det->bbox[2] * det->bbox[2] * anchors[2 * k];
            det->bbox[3] = 2.0f * Logist(curInput[idx + k * info_len_i * total_grid + 3 * total_grid]);
            det->bbox[3] = det->bbox[3] * det->bbox[3] * anchors[2 * k + 1];
            det->conf = box_prob * max_cls_prob;
            det->class_id = class_id;
        }
    }

    void YoloLayerPlugin::forwardGpu(const float* const* inputs, float *output, hipStream_t stream, int batchSize)
    {
        int outputElem = 1 + mMaxOutObject * sizeof(Detection) / sizeof(float);
        for (int idx = 0; idx < batchSize; ++idx) {
            CUDA_CHECK(hipMemsetAsync(output + idx * outputElem, 0, sizeof(float), stream));
        }
        int numElem = 0;
        for (unsigned int i = 0; i < mYoloKernel.size(); ++i) {
            const auto& yolo = mYoloKernel[i];
            numElem = yolo.width * yolo.height * batchSize;
            if (numElem < mThreadCount) mThreadCount = numElem;

            //printf("Net: %d  %d \n", mYoloV5NetWidth, mYoloV5NetHeight);
            CalDetection << < (numElem + mThreadCount - 1) / mThreadCount, mThreadCount, 0, stream >> >
                (inputs[i], output, numElem, mYoloV5NetWidth, mYoloV5NetHeight, mMaxOutObject, yolo.width, yolo.height, (float*)mAnchor[i], mClassCount, outputElem);
        }
    }


    int YoloLayerPlugin::enqueue(int batchSize, const void* const* inputs, void* TRT_CONST_ENQUEUE* outputs, void* workspace, hipStream_t stream) TRT_NOEXCEPT
    {
        forwardGpu((const float* const*)inputs, (float*)outputs[0], stream, batchSize);
        return 0;
    }

    PluginFieldCollection YoloPluginCreator::mFC{};
    std::vector<PluginField> YoloPluginCreator::mPluginAttributes;

    YoloPluginCreator::YoloPluginCreator()
    {
        mPluginAttributes.clear();

        mFC.nbFields = mPluginAttributes.size();
        mFC.fields = mPluginAttributes.data();
    }

    const char* YoloPluginCreator::getPluginName() const TRT_NOEXCEPT
    {
        return "YoloLayer_TRT";
    }

    const char* YoloPluginCreator::getPluginVersion() const TRT_NOEXCEPT
    {
        return "1";
    }

    const PluginFieldCollection* YoloPluginCreator::getFieldNames() TRT_NOEXCEPT
    {
        return &mFC;
    }

    IPluginV2IOExt* YoloPluginCreator::createPlugin(const char* name, const PluginFieldCollection* fc) TRT_NOEXCEPT
    {
        assert(fc->nbFields == 2);
        assert(strcmp(fc->fields[0].name, "netinfo") == 0);
        assert(strcmp(fc->fields[1].name, "kernels") == 0);
        int *p_netinfo = (int*)(fc->fields[0].data);
        int class_count = p_netinfo[0];
        int input_w = p_netinfo[1];
        int input_h = p_netinfo[2];
        int max_output_object_count = p_netinfo[3];
        std::vector<Yolo::YoloKernel> kernels(fc->fields[1].length);
        memcpy(&kernels[0], fc->fields[1].data, kernels.size() * sizeof(Yolo::YoloKernel));
        YoloLayerPlugin* obj = new YoloLayerPlugin(class_count, input_w, input_h, max_output_object_count, kernels);
        obj->setPluginNamespace(mNamespace.c_str());
        return obj;
    }

    IPluginV2IOExt* YoloPluginCreator::deserializePlugin(const char* name, const void* serialData, size_t serialLength) TRT_NOEXCEPT
    {
        // This object will be deleted when the network is destroyed, which will
        // call YoloLayerPlugin::destroy()
        YoloLayerPlugin* obj = new YoloLayerPlugin(serialData, serialLength);
        obj->setPluginNamespace(mNamespace.c_str());
        return obj;
    }
}

